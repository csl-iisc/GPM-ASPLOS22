#include "hip/hip_runtime.h"
#include "srad.h"
#include <stdio.h>

__global__ void
srad_cuda_1(
		  float *E_C, 
		  float *W_C, 
		  float *N_C, 
		  float *S_C,
		  float * J_cuda, 
		  float * C_cuda, 
		  long long cols, 
		  long long rows, 
		  float q0sqr
) 
{

  //block id
  long long bx = blockIdx.x;
  long long by = blockIdx.y;

  //thread id
  long long tx = threadIdx.x;
  long long ty = threadIdx.y;
  
  //indices
  long long index   = cols * BLOCK_SIZE * by + BLOCK_SIZE * bx + cols * ty + tx;
    if (index < cols * rows) {
  long long index_n = cols * BLOCK_SIZE * by + BLOCK_SIZE * bx + tx - cols;
  long long index_s = cols * BLOCK_SIZE * by + BLOCK_SIZE * bx + cols * BLOCK_SIZE + tx;
  long long index_w = cols * BLOCK_SIZE * by + BLOCK_SIZE * bx + cols * ty - 1;
  long long index_e = cols * BLOCK_SIZE * by + BLOCK_SIZE * bx + cols * ty + BLOCK_SIZE;

  float n, w, e, s, jc, g2, l, num, den, qsqr, c;

  //shared memory allocation
  __shared__ float temp[BLOCK_SIZE][BLOCK_SIZE];
  __shared__ float temp_result[BLOCK_SIZE][BLOCK_SIZE];

  __shared__ float north[BLOCK_SIZE][BLOCK_SIZE];
  __shared__ float south[BLOCK_SIZE][BLOCK_SIZE];
  __shared__ float  east[BLOCK_SIZE][BLOCK_SIZE];
  __shared__ float  west[BLOCK_SIZE][BLOCK_SIZE];

        //load data to shared memory
        if ( by == 0 ){
            north[ty][tx] = J_cuda[BLOCK_SIZE * bx + tx]; 
            south[ty][tx] = J_cuda[index_s];
        }
        else if ( by == gridDim.y - 1 ){
            north[ty][tx] = J_cuda[index_n]; 
            south[ty][tx] = J_cuda[cols * BLOCK_SIZE * (gridDim.y - 1) + BLOCK_SIZE * bx + cols * ( BLOCK_SIZE - 1 ) + tx];
        }
        else {
            north[ty][tx] = J_cuda[index_n]; 
            south[ty][tx] = J_cuda[index_s];
        }
        __syncthreads();

        if ( bx == 0 ){
            west[ty][tx] = J_cuda[cols * BLOCK_SIZE * by + cols * ty];
            east[ty][tx] = J_cuda[index_e]; 
        }
        else if ( bx == gridDim.x - 1 ){
            west[ty][tx] = J_cuda[index_w];
            east[ty][tx] = J_cuda[cols * BLOCK_SIZE * by + BLOCK_SIZE * ( gridDim.x - 1) + cols * ty + BLOCK_SIZE-1];
        }
        else {
            west[ty][tx] = J_cuda[index_w];
            east[ty][tx] = J_cuda[index_e];
        }

        __syncthreads();

        temp[ty][tx] = J_cuda[index];

        __syncthreads();
   jc = temp[ty][tx];

   if ( ty == 0 && tx == 0 ){ //nw
	n  = north[ty][tx] - jc;
    s  = temp[ty+1][tx] - jc;
    w  = west[ty][tx]  - jc; 
    e  = temp[ty][tx+1] - jc;
   }	    
   else if ( ty == 0 && tx == BLOCK_SIZE-1 ){ //ne
	n  = north[ty][tx] - jc;
    s  = temp[ty+1][tx] - jc;
    w  = temp[ty][tx-1] - jc; 
    e  = east[ty][tx] - jc;
   }
   else if ( ty == BLOCK_SIZE -1 && tx == BLOCK_SIZE - 1){ //se
	n  = temp[ty-1][tx] - jc;
    s  = south[ty][tx] - jc;
    w  = temp[ty][tx-1] - jc; 
    e  = east[ty][tx]  - jc;
   }
   else if ( ty == BLOCK_SIZE -1 && tx == 0 ){//sw
	n  = temp[ty-1][tx] - jc;
    s  = south[ty][tx] - jc;
    w  = west[ty][tx]  - jc; 
    e  = temp[ty][tx+1] - jc;
   }

   else if ( ty == 0 ){ //n
	n  = north[ty][tx] - jc;
    s  = temp[ty+1][tx] - jc;
    w  = temp[ty][tx-1] - jc; 
    e  = temp[ty][tx+1] - jc;
   }
   else if ( tx == BLOCK_SIZE -1 ){ //e
	n  = temp[ty-1][tx] - jc;
    s  = temp[ty+1][tx] - jc;
    w  = temp[ty][tx-1] - jc; 
    e  = east[ty][tx] - jc;
   }
   else if ( ty == BLOCK_SIZE -1){ //s
	n  = temp[ty-1][tx] - jc;
    s  = south[ty][tx] - jc;
    w  = temp[ty][tx-1] - jc; 
    e  = temp[ty][tx+1] - jc;
   }
   else if ( tx == 0 ){ //w
	n  = temp[ty-1][tx] - jc;
    s  = temp[ty+1][tx] - jc;
    w  = west[ty][tx] - jc; 
    e  = temp[ty][tx+1] - jc;
   }
   else{  //the data elements which are not on the borders 
	n  = temp[ty-1][tx] - jc;
    s  = temp[ty+1][tx] - jc;
    w  = temp[ty][tx-1] - jc; 
    e  = temp[ty][tx+1] - jc;
   }


    g2 = ( n * n + s * s + w * w + e * e ) / (jc * jc);

    l = ( n + s + w + e ) / jc;

	num  = (0.5*g2) - ((1.0/16.0)*(l*l)) ;
	den  = 1 + (.25*l);
	qsqr = num/(den*den);

	// diffusion coefficent (equ 33)
	den = (qsqr-q0sqr) / (q0sqr * (1+q0sqr)) ;
	c = 1.0 / (1.0+den) ;

        // saturate diffusion coefficent
	    if (c < 0){temp_result[ty][tx] = 0;}
	    else if (c > 1) {temp_result[ty][tx] = 1;}
	    else {temp_result[ty][tx] = c;}

            //__syncthreads(); // Seems unneeded?

        C_cuda[index] = temp_result[ty][tx];
	    E_C[index] = e;
	    W_C[index] = w;
	    S_C[index] = s;
	    N_C[index] = n;
    }
}

__global__ void
srad_cuda_2(
		  float *E_C, 
		  float *W_C, 
		  float *N_C, 
		  float *S_C,	
		  float * J_cuda, 
		  float * C_cuda, 
		  long long cols, 
		  long long rows, 
		  float lambda,
		  float q0sqr
) 
{
	//block id
	long long bx = blockIdx.x;
    long long by = blockIdx.y;

	//thread id
    long long tx = threadIdx.x;
    long long ty = threadIdx.y;

	//indices
    long long index   = cols * BLOCK_SIZE * by + BLOCK_SIZE * bx + cols * ty + tx;
    if (index < cols * rows)
    {
        long long index_s = cols * BLOCK_SIZE * by + BLOCK_SIZE * bx + cols * BLOCK_SIZE + tx;
        long long index_e = cols * BLOCK_SIZE * by + BLOCK_SIZE * bx + cols * ty + BLOCK_SIZE;
        float cc, cn, cs, ce, cw, d_sum;

	//shared memory allocation
	__shared__ float south_c[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float  east_c[BLOCK_SIZE][BLOCK_SIZE];

    __shared__ float c_cuda_temp[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float c_cuda_result[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float temp[BLOCK_SIZE][BLOCK_SIZE];

    //load data to shared memory
	temp[ty][tx]      = J_cuda[index];

    __syncthreads();

	if ( by == gridDim.y - 1 ){
	south_c[ty][tx] = C_cuda[cols * BLOCK_SIZE * (gridDim.y - 1) + BLOCK_SIZE * bx + cols * ( BLOCK_SIZE - 1 ) + tx];
	}
    else {
        south_c[ty][tx] = C_cuda[index_s];
    }
	__syncthreads();

	if ( bx == gridDim.x - 1 ){
	east_c[ty][tx] = C_cuda[cols * BLOCK_SIZE * by + BLOCK_SIZE * ( gridDim.x - 1) + cols * ty + BLOCK_SIZE-1];
	}
    else {
        east_c[ty][tx] = C_cuda[index_e];
	 }

    __syncthreads();
  
    c_cuda_temp[ty][tx]      = C_cuda[index];

    __syncthreads();

	cc = c_cuda_temp[ty][tx];

       if ( ty == BLOCK_SIZE -1 && tx == BLOCK_SIZE - 1){ //se
	    cn  = cc;
        cs  = south_c[ty][tx];
        cw  = cc; 
        ce  = east_c[ty][tx];
       } 
       else if ( tx == BLOCK_SIZE -1 ){ //e
	    cn  = cc;
        cs  = c_cuda_temp[ty+1][tx];
        cw  = cc; 
        ce  = east_c[ty][tx];
       }
       else if ( ty == BLOCK_SIZE -1){ //s
	    cn  = cc;
        cs  = south_c[ty][tx];
        cw  = cc; 
        ce  = c_cuda_temp[ty][tx+1];
       }
       else{ //the data elements which are not on the borders 
	    cn  = cc;
        cs  = c_cuda_temp[ty+1][tx];
        cw  = cc; 
        ce  = c_cuda_temp[ty][tx+1];
       }

       // divergence (equ 58)
       d_sum = cn * N_C[index] + cs * S_C[index] + cw * W_C[index] + ce * E_C[index];

       // image update (equ 61)
       c_cuda_result[ty][tx] = temp[ty][tx] + 0.25 * lambda * d_sum;

       //__syncthreads(); // Seems unneeded?
                  
       J_cuda[index] = c_cuda_result[ty][tx];
    }
}
