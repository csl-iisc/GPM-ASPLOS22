#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2016 University of Cordoba and University of Illinois
 * All rights reserved.
 *
 * Developed by:    IMPACT Research Group
 *                  University of Cordoba and University of Illinois
 *                  http://impact.crhc.illinois.edu/
 *
 * Permission is hereby granted, free of charge, to any person obtaining a copy
 * of this software and associated documentation files (the "Software"), to deal
 * with the Software without restriction, including without limitation the 
 * rights to use, copy, modify, merge, publish, distribute, sublicense, and/or
 * sell copies of the Software, and to permit persons to whom the Software is
 * furnished to do so, subject to the following conditions:
 *
 *      > Redistributions of source code must retain the above copyright notice,
 *        this list of conditions and the following disclaimers.
 *      > Redistributions in binary form must reproduce the above copyright
 *        notice, this list of conditions and the following disclaimers in the
 *        documentation and/or other materials provided with the distribution.
 *      > Neither the names of IMPACT Research Group, University of Cordoba, 
 *        University of Illinois nor the names of its contributors may be used 
 *        to endorse or promote products derived from this Software without 
 *        specific prior written permission.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE 
 * CONTRIBUTORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
 * OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS WITH
 * THE SOFTWARE.
 *
 */

#define _CUDA_COMPILER_

#include "support/common.h"

// CUDA kernel ------------------------------------------------------------------------------------------
__global__ void BFS_gpu(Node *graph_nodes_av, Edge *graph_edges_av, int *cost,
    int *color, int *q1, int *q2, int *n_t,
    int *head, int *tail, int *threads_end, int *threads_run,
    int *overflow, int *iter, int LIMIT, const int CPU) {

    extern __shared__ int l_mem[];
    int* tail_bin = l_mem;
    int* l_q2 = (int*)&tail_bin[1];
    int* shift = (int*)&l_q2[W_QUEUE_SIZE];
    int* base = (int*)&shift[1];

    const int tid     = threadIdx.x;
    const int gtid    = blockIdx.x * blockDim.x + threadIdx.x;
    const int MAXWG   = gridDim.x;
    const int WG_SIZE = blockDim.x;

    int iter_local = atomicAdd(&iter[0], 0);

    int n_t_local = atomicAdd(n_t, 0);

    if(tid == 0) {
        // Reset queue
        *tail_bin = 0;
    }

    // Fetch frontier elements from the queue
    if(tid == 0)
        *base = atomicAdd(&head[0], WG_SIZE);
    __syncthreads();

    int my_base = *base;
    while(my_base < n_t_local) {
        if(my_base + tid < n_t_local && *overflow == 0) {
            // Visit a node from the current frontier
            int pid = q1[my_base + tid];
            //////////////// Visit node ///////////////////////////
            atomicExch(&cost[pid], iter_local); // Node visited
            Node cur_node;
            cur_node.x = graph_nodes_av[pid].x;
            cur_node.y = graph_nodes_av[pid].y;
            // For each outgoing edge
            for(int i = cur_node.x; i < cur_node.y + cur_node.x; i++) {
                int id        = graph_edges_av[i].x;
                int old_color = atomicMin(&color[id], iter_local);
                if(old_color > iter_local) {
                    // Push to the queue
                    int tail_index = atomicAdd(tail_bin, 1);
                    if(tail_index >= W_QUEUE_SIZE) {
                        *overflow = 1;
                    } else
                        l_q2[tail_index] = id;
                }
            }
        }
        if(tid == 0)
            *base = atomicAdd(&head[0], WG_SIZE); // Fetch more frontier elements from the queue
        __syncthreads();
        my_base = *base;
    }
    /////////////////////////////////////////////////////////
    // Compute size of the output and allocate space in the global queue
    if(tid == 0) {
        *shift = atomicAdd(&tail[0], *tail_bin);
    }
    __syncthreads();
    ///////////////////// CONCATENATE INTO GLOBAL MEMORY /////////////////////
    int local_shift = tid;
    while(local_shift < *tail_bin) {
        q2[*shift + local_shift] = l_q2[local_shift];
        // Multiple threads are copying elements at the same time, so we shift by multiple elements for next iteration
        local_shift += WG_SIZE;
    }
    //////////////////////////////////////////////////////////////////////////

    if(gtid == 0) {
        atomicAdd(&iter[0], 1);
    }
}

hipError_t call_BFS_gpu(int blocks, int threads, Node *graph_nodes_av, Edge *graph_edges_av, int *cost,
    int *color, int *q1, int *q2, int *n_t,
    int *head, int *tail, int *threads_end, int *threads_run,
    int *overflow, int *iter, int LIMIT, const int CPU, int l_mem_size){

    dim3 dimGrid(blocks);
    dim3 dimBlock(threads);
    BFS_gpu<<<dimGrid, dimBlock, l_mem_size>>>(graph_nodes_av, graph_edges_av, cost,
        color, q1, q2, n_t,
        head, tail, threads_end, threads_run,
        overflow, iter, LIMIT, CPU);
    
    hipError_t err = hipGetLastError();
    return err;
}
