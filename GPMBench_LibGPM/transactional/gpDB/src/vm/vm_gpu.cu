#include "hip/hip_runtime.h"
// vim: filetype=c

#include <hip/hip_runtime.h>
#include <fcntl.h>

/**
 * @file
 * @section Description 
 *
 * This file contains the virg_vm_gpu() function and virginia_gpu(), the cuda
 * kernel virtual machine. This is the only file that is compiled with nvcc,
 * rather than gcc or icc.
 */

extern "C" {
#include "virginian.h"
}

// defining VIRG_NOTWOSTEP disables the two step result writing procedure that
// is efficent for mapped memory
#ifdef VIRG_NOTWOSTEP
#define VIRG_NOTWOSTEP 1
#else
#define VIRG_NOTWOSTEP 0
#endif

/// Used in testing to return size test array from gpu-compiled code
const size_t *virg_gpu_getsizes()
{
	return &virg_testsizes[0];
}

/// GPU constant memory variable to hold the virtual machine execution context
__constant__ virg_vm vm;

/** 
 * GPU constant memory array to hold the tablet meta information for each of
 * the tablet slots allocated on the GPU
 */
__constant__ virg_tablet_meta meta[VIRG_GPU_TABLETS];

/// total number of result rows output during mapped memory vm execution
__device__ unsigned row_counter;

/// total number of result rows that have reached global memory 
__device__ unsigned rowbuff_counter;

/// counter incremented by threadblocks when they write
__device__ unsigned threadblock_order;

__global__ void virginia_gpu(unsigned tab_slot, unsigned res_slot, void* tab_,
	void* res_, unsigned start_row, unsigned num_rows, void *scratch);

/**
 * @ingroup vm
 * @brief Execute the data-parallel portion of an opcode program on a GPU
 *
 * This function handles setting up and launching the virtual machine kernel
 * that executes the virtual machine on the GPU. The user has a choice between
 * serial kernel executions for each tablet, streaming executions with
 * overlapping memcpys, and mapped execution, which is currently the fastest
 * option.
 *
 * If streaming and memory mapping are both disabled in the virginian struct,
 * then kernels will be processed on the GPU serially. This means that the data
 * tablet will be transferred to GPU memory, the virtual machine executed, then
 * the result tablet transferred back for each tablet with no overlap. This is
 * the slowest method of GPU execution, and it does not require pinned memory.
 *
 * If streaming is enabled, regardless of the mapped memory setting, then it is
 * used. This works by allocating a fixed number of tablet streams, set equal to
 * half of the number of allocated GPU tablet slots, and overlapping data
 * transfer, kernel execution, and result transfer for tablets. Each stream gets
 * a data and result tablet slot on the GPU, and the loop iterates through each
 * stream in a round-robin fashion executing virtual machines on tablets. Note
 * that when we loop back and re-use streams then we block to wait for execution
 * if the previous asynchronous launches. On Tesla C1060 hardware you cannot
 * overlap transfers to the GPU with transfers from it, and thus streaming is
 * about as fast as serial execution. Also note that on this hardware the
 * semantics for asynchronous operations are somewhat frusterating. Based on
 * informal tests, it appears that even though asynchronous operations are
 * completely asynchronous with respect to the calling thread, the order in
 * which they are added to streams has an effect on when they are run. Thus,
 * there are non-optimal orderings of memory copies and kernel executions when
 * multiple asynchronous operations are queued in multiple streams. Even with
 * smarter ordering, however, the inability of current hardware to transfer both
 * ways across the PCI bus simultaneously means that this method of execution is
 * currently only useful if either the data or results of a query are resident
 * in GPU memory for the entire processing operation. For some reason I could
 * only create 6 streams at a time during my testing, so you may encounter
 * problems if you allocate more than 12 gpu tablet slots.
 *
 * Mapped memory is currently the fastest method for GPU processing by far. It
 * is also much simpler, since we have fewer memory copies and we don't have to
 * manage multiple streams at the same time. The tablet row counter is
 * implemented as a separate variable for mapped memory, since it would be very
 * very expensive to perform atomic operations on a mapped location.
 *
 * @param v     Pointer to the state struct of the database system
 * @param vm	Pointer to the context struct of the virtual machine
 * @param tab	Pointer to the pointer to the current data tablet to process
 * @param res	Pointer to the pointer to the current result tablet
 * @param num_tablets Number of tablets to process on the GPU, 0 if as many as
 * possible
 * @return VIRG_SUCCESS or VIRG_FAIL depending on errors during the function
 * call
 */
int virg_vm_gpu(virginian *v, virg_vm *vm_, virg_tablet_meta **tab, virg_tablet_meta **res, unsigned num_tablets, void *output, unsigned *get_rows)
{
	unsigned proced = 0;
	//num_tablets = 5;

	VIRG_CHECK(v->threads_per_block != VIRG_THREADSPERBLOCK,
		"Cannot change compile-time threads per block");
	VIRG_CHECK(v->threads_per_block == v->threads_per_block & 0xFFFFFFC0,
		"Threads per block must be a multiple of 64");

	// execute GPU kernels in serial with no overlapping memory copies
	if(v->use_stream == 0 && v->use_mmap == 0)
	{
		// copy virtual machine context to constant memory
		hipMemcpyToSymbol(HIP_SYMBOL(vm), (char*)vm_,
			sizeof(virg_vm), 0, hipMemcpyHostToDevice);
		VIRG_CUDCHK("serial const memcpy 1");

		// copy result meta information to constant memory
		// we only need to do this once for multiple kernel calls because the
		// information about the column spacing is identical between all result
		// tablets
		hipMemcpyToSymbol(HIP_SYMBOL(meta), (char*)res[0],
			sizeof(virg_tablet_meta), sizeof(virg_tablet_meta),
			hipMemcpyHostToDevice);
		VIRG_CUDCHK("serial const memcpy 2");

		void *tab_slot = v->gpu_slots;
		void *res_slot = (char*)v->gpu_slots + VIRG_TABLET_SIZE;

		// create timers
		hipEvent_t start, data, kernel, results;
		hipEventCreate(&start);
		hipEventCreate(&data);
		hipEventCreate(&kernel);
		hipEventCreate(&results);
		vm_->timing1 = 0;
		vm_->timing2 = 0;
		vm_->timing3 = 0;
		virg_tablet_meta *dummy_res;
		hipMalloc((void**)&dummy_res, VIRG_TABLET_SIZE);
		virg_tablet_meta *metadata;
		hipHostMalloc((void**)&metadata, sizeof(virg_tablet_meta));
		size_t offset = 0;

		while(1) {
			VIRG_CUDCHK("const clear");

			VIRG_CUDCHK("before serial const 2 memcpy");
			// copy data tablet meta information to constant memory
			hipMemcpyToSymbol(HIP_SYMBOL(meta), (char*)tab[0],
				sizeof(virg_tablet_meta), 0, hipMemcpyHostToDevice);
			VIRG_CUDCHK("serial const 2 memcpy");

			// round threadblocks up given number of rows to process and threads
			// per block
			unsigned rows = tab[0]->rows;
			int blocks = (rows + v->threads_per_block - 1) /
				v->threads_per_block;
			blocks = max(blocks, 1);
			assert(blocks < 65536);

			// start timer
			hipEventRecord(start, 0);

			// copy entire data tablet to GPU memory
			/*hipMemcpy(tab_slot, (char*)tab[0]->base_ptr,
				tab[0]->size, hipMemcpyDeviceToDevice);
			hipMemcpy(tab_slot, (char*)tab[0],
				sizeof(virg_tablet_meta), hipMemcpyHostToDevice);
			// copy res meta information to GPU memory, where the rows element
			// will be updated as result rows are output
			hipMemcpy(res_slot, (char*)res[0],
				sizeof(virg_tablet_meta), hipMemcpyHostToDevice);
			VIRG_CUDCHK("data memcpy");*/

#ifdef VIRG_DEBUG
			hipMemset((char*)res_slot + sizeof(virg_tablet_meta),
				0xDEADBEEF, VIRG_TABLET_SIZE - sizeof(virg_tablet_meta));
#endif

			// record we're done with data transfer
			hipEventRecord(data, 0);

			virg_timer_start();

			// kernel launch
			void* tab_arg = v->gpu_slots;
			void* res_arg = (char*)v->gpu_slots + VIRG_TABLET_SIZE;
			if(output == NULL) {
				hipMemcpy(dummy_res, res[0]->base_ptr, VIRG_TABLET_SIZE, hipMemcpyDeviceToDevice);
				hipMemcpy(dummy_res, res[0], sizeof(virg_tablet_meta), hipMemcpyHostToDevice);
				hipMemcpy(&dummy_res->base_ptr, &dummy_res, sizeof(virg_tablet_meta*), hipMemcpyHostToDevice);
			}
			else {
				hipMemcpy(dummy_res, res[0], sizeof(virg_tablet_meta), hipMemcpyHostToDevice);
				hipMemset(&dummy_res->rows, 0, sizeof(unsigned));
				hipMemcpy(&dummy_res->base_ptr, &dummy_res, sizeof(void *), hipMemcpyHostToDevice);
			}
			VIRG_CUDCHK("Single kernel launch");

			virginia_gpu<<<blocks, v->threads_per_block>>>
				(0, 1, tab[0]->base_ptr, dummy_res, 0, 0, NULL);
	hipDeviceSynchronize();

			unsigned num_rows;
			hipMemcpy(&num_rows, &dummy_res->rows, sizeof(unsigned), hipMemcpyDeviceToHost);
			//printf("%u %u %u %u\n", num_rows, res[0]->rows, res[0]->rows, res[0]->possible_rows);
			// if the result tablet is full

			if(output == NULL) {
				if(num_rows >= res[0]->possible_rows) {
					// unlock current tablet and allocate another one
					virg_tablet_unlock(v, res[0]->id);
					virg_vm_allocresult(v, vm_, res, res[0]);
					virg_tablet_lock(v, res[0]->id);
					continue;
				}
				VIRG_CUDCHK("Single kernel launch");
				if(num_rows != res[0]->rows) {
					void *b_ptr = res[0]->base_ptr;
					hipMemcpy(res[0]->base_ptr, dummy_res, VIRG_TABLET_SIZE, hipMemcpyDeviceToDevice);
					hipMemcpy(res[0], dummy_res, sizeof(virg_tablet_meta), hipMemcpyDeviceToHost);
					res[0]->base_ptr = b_ptr;
				}
			}
			else {
				hipMemcpy(metadata, dummy_res, sizeof(virg_tablet_meta), hipMemcpyDeviceToHost);
				if(get_rows != NULL)
					*get_rows += metadata->rows;
				char *fixed = (char*)dummy_res + metadata->fixed_block;
				// get columns one by one and place entire row in reader buffer
				for(int i = 0; i < metadata->fixed_columns; i++) {
					size_t stride = metadata->fixed_stride[i];
					void *src = fixed + metadata->fixed_offset[i];
					hipMemcpy((char*)output + offset, src, metadata->rows * stride, hipMemcpyDeviceToHost);
					offset += metadata->rows * stride;
				}
			}

			VIRG_CUDCHK("Single kernel launch");

			// record we're done with the kernel call
			hipEventRecord(kernel, 0);

			// transfer result tablet back from GPU memory
			/*hipMemcpy((char*)res[0]->base_ptr, res_slot,
				VIRG_TABLET_SIZE, hipMemcpyDeviceToDevice);
			void *p = res[0]->base_ptr;
			hipMemcpy((char*)res[0], res_slot,
				sizeof(virg_tablet_meta), hipMemcpyDeviceToHost);
			res[0]->base_ptr = p;*/
			//virg_print_tablet_meta(res[0]);

			// record we're done with the results transfer
			hipEventRecord(results, 0);

			// output timing results for this tablet
			float f[3];
			hipEventElapsedTime(&f[0], start, data);
			hipEventElapsedTime(&f[1], data, kernel);
			hipEventSynchronize(results);
			hipEventElapsedTime(&f[2], kernel, results);
			//fprintf(stderr, "serial block %u: %f %f %f %f\n", proced, cum, f[0], f[1], f[2]);
			vm_->timing1 += f[0];
			vm_->timing2 += f[1];
			vm_->timing3 += f[2];

			proced++;

			// if we've processed enough tablets exit the loop
			if(tab[0]->last_tablet || (num_tablets != 0 && proced >= num_tablets))
				break;

			// load next data tablet
			virg_db_loadnext(v, tab);
			// if this tablet has no rows, break from this loop
			// this occurs when a new data tablet is created during an insert
			// operation but no rows have been added to it yet
			if(tab[0]->rows == 0)
				break;

			// safely allocate next result tablet
			//virg_tablet_meta *temp = res[0];
			//virg_vm_allocresult(v, vm_, res, res[0]);
			//virg_tablet_unlock(v, temp->id);
		}

		vm_->timing1 /= 1000;
		vm_->timing2 /= 1000;
		vm_->timing3 /= 1000;

		// destruct timers
		hipEventDestroy(start);
		hipEventDestroy(data);
		hipEventDestroy(kernel);
		hipEventDestroy(results);
	}
	// if the streaming functionality is turned on
	else if(v->use_stream)
	{
		// copy virtual machine context to GPU constant memory
		hipMemcpyToSymbol(HIP_SYMBOL(vm), (char*)vm_,
			sizeof(virg_vm), 0, hipMemcpyHostToDevice);
		VIRG_CUDCHK("const memcpy");

		// we should always have an even number of tablets
		assert(VIRG_GPU_TABLETS % 2 == 0);
		unsigned stream_width = VIRG_GPU_TABLETS / 2;
		hipStream_t stream[stream_width];
		unsigned slot_ids[stream_width];
		int slot_wait = 0;
		unsigned i;

		// construct streams 
		for(i = 0; i < stream_width; i++)
			hipStreamCreate(&stream[i]);
		VIRG_CUDCHK("stream create");

		// create timers for each stream independently
		hipEvent_t ev_create[stream_width], ev_start[stream_width],
				ev_data[stream_width], ev_kernel[stream_width],
				ev_results[stream_width];
		for(i = 0; i < stream_width; i++) {
			hipEventCreate(&ev_create[i]);
			hipEventCreate(&ev_start[i]);
			hipEventCreate(&ev_data[i]);
			hipEventCreate(&ev_kernel[i]);
			hipEventCreate(&ev_results[i]);
		}

		// start timer for each stream
		for(i = 0; i < stream_width; i++)
			hipEventRecord(ev_create[i], stream[i]);

		// process tablets until finished
		for(i = 0; 1; i++) {
			// if every stream has been used, go back to use the first stream
			// again
			if(i >= stream_width) {
				i = 0;
				slot_wait = 1;
			}

			// if we are re-using streams then we need to block until they are
			// actually finished
			if(slot_wait) {
				VIRG_CUDCHK("before stream synchronize");
				// block
				hipStreamSynchronize(stream[i]);
				VIRG_CUDCHK("stream synchronize");
				// unlock the tablets that the stream was using
				virg_tablet_unlock(v, slot_ids[i * 2]);
				virg_tablet_unlock(v, slot_ids[i * 2 + 1]);

				// record processing completion and output times
				hipEventSynchronize(ev_results[i]);
				float f[4];
				hipEventElapsedTime(&f[0], ev_create[i], ev_start[i]);
				hipEventElapsedTime(&f[1], ev_start[i], ev_data[i]);
				hipEventElapsedTime(&f[2], ev_data[i], ev_kernel[i]);
				hipEventElapsedTime(&f[3], ev_kernel[i], ev_results[i]);
				fprintf(stderr, "stream %u: %f %f %f %f\n", i, f[0], f[1], f[2], f[3]);
			}

			// if the data tablet doesn't have any rows then we're finished
			if(tab[0]->rows == 0) {
				proced++;
				slot_ids[i * 2] = tab[0]->id;
				slot_ids[i * 2 + 1] = res[0]->id;
				break;
			}

#ifdef VIRG_DEBUG
			virg_tablet_check(tab[0]);
#endif

			// round up blocks given rows to process and threads per block	
			int blocks = (tab[0]->rows + v->threads_per_block - 1) / v->threads_per_block;
			assert(blocks < 65536);

			virg_tablet_meta *temp_tab, *temp_res;

			// if there are still tablets to process load, otherwise don't
			// note that we don't exit here because we need to wait for the
			// other streams to finish
			if(!tab[0]->last_tablet && !(num_tablets != 0 && proced + 1 > num_tablets)) {
				virg_db_load(v, tab[0]->next, &temp_tab);
				virg_vm_allocresult(v, vm_, &temp_res, res[0]);
			}

			// start timer for this stream
			hipEventRecord(ev_start[i], stream[i]);

			// start tablet memcpy for this stream
			hipMemcpyAsync((char*)v->gpu_slots + (i * 2) * VIRG_TABLET_SIZE,
				(char*)tab[0]->base_ptr, tab[0]->size, hipMemcpyDeviceToDevice, stream[i]);
			//virg_print_tablet_meta(tab[0]);
			VIRG_CUDCHK("tab memcpy");

			// start tablet meta to constant memory memcpy for this stream
			hipMemcpyToSymbolAsync(HIP_SYMBOL(meta), (char*)tab[0],
				sizeof(virg_tablet_meta), i * 2 * sizeof(virg_tablet_meta),
				hipMemcpyHostToDevice, stream[i]);
			VIRG_CUDCHK("tab meta");

			// if we haven't put the result meta information in this stream's
			// constant memory area yet
			if(!slot_wait) {
				// copy result meta information for this stream
				hipMemcpyToSymbolAsync(HIP_SYMBOL(meta), (char*)res[0],
					sizeof(virg_tablet_meta), (i * 2 + 1) * sizeof(virg_tablet_meta),
					hipMemcpyHostToDevice, stream[i]);
				VIRG_CUDCHK("res meta");
			}

			// copy meta information to global memory as well so that the rows
			// variable can be updated during query execution
			hipMemcpyAsync((char*)v->gpu_slots + (i * 2 + 1) * VIRG_TABLET_SIZE,
				(char*)res[0], sizeof(virg_tablet_meta), hipMemcpyHostToDevice, stream[i]);
			VIRG_CUDCHK("res setup memcpy");

			// record we're done with data transfer for this stream
			hipEventRecord(ev_data[i], stream[i]);

			// launch the kernel for this stream
			void *tab_arg = (char*)v->gpu_slots + (i * 2) * VIRG_TABLET_SIZE;
			void *res_arg = (char*)v->gpu_slots + (i * 2 + 1) * VIRG_TABLET_SIZE;
			virginia_gpu<<<blocks, v->threads_per_block, 0, stream[i]>>>
				(i * 2, i * 2 + 1, tab_arg, res_arg, 0, 0, NULL);
			VIRG_CUDCHK("kernel");

			// record that the kernel execution has finished
			hipEventRecord(ev_kernel[i], stream[i]);

			// copy result tablet back for this stream
			hipMemcpyAsync((char*)res[0]->base_ptr, (char*)v->gpu_slots + (i * 2 + 1) * VIRG_TABLET_SIZE,
				VIRG_TABLET_SIZE, hipMemcpyDeviceToDevice, stream[i]);
			hipMemcpyAsync((char*)res[0], (char*)v->gpu_slots + (i * 2 + 1) * VIRG_TABLET_SIZE,
				sizeof(virg_tablet_meta), hipMemcpyDeviceToDevice, stream[i]);
			VIRG_CUDCHK("res memcpy");

			// record that we're done with the result transfer for this stream
			hipEventRecord(ev_results[i], stream[i]);

			// store the current data and result tablet pointers in the stream's
			// slot
			proced++;
			slot_ids[i * 2] = tab[0]->id;
			slot_ids[i * 2 + 1] = res[0]->id;

			// check if we've processed enough tablets
			if(tab[0]->last_tablet || (num_tablets != 0 && proced >= num_tablets))
				break;

			tab[0] = temp_tab;
			res[0] = temp_res;
		}

		i++;

		unsigned j;
		if(!slot_wait)
			i = 0;
		// for each unfinished stream
		for(j = 0; j < VIRG_MIN(stream_width, proced); j++, i++) {
			if(i >= stream_width)
				i = 0;

			// wait for the stream to finish and print timing information
			hipStreamSynchronize(stream[i]);
			hipEventSynchronize(ev_results[i]);
			float f[4];
			hipEventElapsedTime(&f[0], ev_create[i], ev_start[i]);
			hipEventElapsedTime(&f[1], ev_start[i], ev_data[i]);
			hipEventElapsedTime(&f[2], ev_data[i], ev_kernel[i]);
			hipEventElapsedTime(&f[3], ev_kernel[i], ev_results[i]);
			//fprintf(stderr, "stream %u: %f %f %f %f\n", i, f[0], f[1], f[2], f[3]);

			// leave last data and result tablets locked
			if(j < stream_width - 1 && j < proced - 1) {
				virg_tablet_unlock(v, slot_ids[i * 2]);
				virg_tablet_unlock(v, slot_ids[i * 2 + 1]);
			}
		}

		// destruct timers and streams
		for(i = 0; i < stream_width; i++) {
			hipEventDestroy(ev_create[i]);
			hipEventDestroy(ev_start[i]);
			hipEventDestroy(ev_data[i]);
			hipEventDestroy(ev_kernel[i]);
			hipEventDestroy(ev_results[i]);
			hipStreamDestroy(stream[i]);
		}
	}
	// memory mapped kernel execution
	else if(v->use_mmap)
	{
		assert(VIRG_GPU_TABLETS >= 2);
#ifdef VIRG_NOPINNED
		VIRG_CHECK(1, "cannot use mapped execution without pinned memory");
#endif

		// copy virtual machine context into gpu constant memory
		hipMemcpyToSymbol(HIP_SYMBOL(vm), (char*)vm_,
			sizeof(virg_vm), 0, hipMemcpyHostToDevice);
		// copy result tablet meta data into gpu constant memory
		// this needs to be done only once since the column sizes etc don't
		// change
		hipMemcpyToSymbol(HIP_SYMBOL(meta), (char*)res[0],
			sizeof(virg_tablet_meta), sizeof(virg_tablet_meta), hipMemcpyHostToDevice);
		VIRG_CUDCHK("mapped const memcpy");

		hipMemcpyToSymbol(HIP_SYMBOL(meta), (char*)tab[0],
			sizeof(virg_tablet_meta), 0, hipMemcpyHostToDevice);
		VIRG_CUDCHK("mapped const 2 memcpy");

		// construct timers
		hipEvent_t start, data, kernel, results;
		hipEventCreate(&start);
		hipEventCreate(&data);
		hipEventCreate(&kernel);
		hipEventCreate(&results);
		float cum = 0;

		while(1) {
			// start timer
			hipEventRecord(start, 0);

			//virg_print_tablet_meta(tab[0]);

			//fprintf(stderr, "::::%u\n", sizeof(virg_tablet_meta));

			// copy tablet meta information to gpu constant memory
			VIRG_CUDCHK("before const 2 memcpy");
			hipMemcpyToSymbol(HIP_SYMBOL(meta), (char*)tab[0],
				sizeof(virg_tablet_meta), 0, hipMemcpyHostToDevice);
			VIRG_CUDCHK("const 2 memcpy");

			// round number of thread blocks up given the number of rows to
			// process and the threads per block
			unsigned rows = tab[0]->rows;
			int blocks = (rows + v->threads_per_block - 1) / v->threads_per_block;
			assert(blocks < 65536);

			unsigned zero = 0;
			// copy 0 into the result row counter
			hipMemcpyToSymbol(HIP_SYMBOL(row_counter), (char*)&zero,
				sizeof(unsigned), 0, hipMemcpyHostToDevice);
			// copy 0 into the result row buffer counter
			hipMemcpyToSymbol(HIP_SYMBOL(rowbuff_counter), (char*)&zero,
				sizeof(unsigned), 0, hipMemcpyHostToDevice);
			hipMemcpyToSymbol(HIP_SYMBOL(threadblock_order), (char*)&zero,
				sizeof(unsigned), 0, hipMemcpyHostToDevice);
			hipMemset((char*)v->gpu_slots + VIRG_TABLET_SIZE, 0,
				sizeof(unsigned) * blocks);
			VIRG_CUDCHK("row_counter set");

			// record that we're done transferring data
			// since we're using mapped memory this is negligible since we just
			// need to set constant memory and 2 variables
			hipEventRecord(data, 0);

			// get gpu pointers to the data and result tablets in main memory
			void *tab_arg;
			void *res_arg;
			hipHostGetDevicePointer(&tab_arg, tab[0], 0);
			VIRG_CUDCHK("get tab device ptr");
			hipHostGetDevicePointer(&res_arg, res[0], 0);
			VIRG_CUDCHK("get res device ptr");

			// launch kernel using mapped pointers
			virginia_gpu<<<blocks, v->threads_per_block>>>
				(0, 1, tab_arg, res_arg, 0, 0, v->gpu_slots);
			VIRG_CUDCHK("Single mapped kernel launch");

			// record we're done with the kernel call
			hipEventRecord(kernel, 0);

			// copy the number of tablet result rows from gpu memory
			hipMemcpyFromSymbol((char*)&res[0]->rows, (char*)&row_counter,
				sizeof(unsigned), 0, hipMemcpyDeviceToHost);

			// record that we're done transferring results information
			// this should also be negligible
			hipEventRecord(results, 0);

			// get timing results
			float f[3];
			hipEventElapsedTime(&f[0], start, data);
			hipEventElapsedTime(&f[1], data, kernel);
			hipEventSynchronize(results);
			hipEventElapsedTime(&f[2], kernel, results);

			// print timing information
			//fprintf(stderr, "block %u: %f %f %f %f\n", proced, cum, f[0], f[1], f[2]);
			
			// add to cumulative time
			cum += f[0] + f[1] + f[2];

			proced++;

			// check if we're done processing tablets
			if(tab[0]->last_tablet || (num_tablets != 0 && proced >= num_tablets))
				break;

			// load next data tablet
			virg_db_loadnext(v, tab);

			// if this data tablet has no rows, finish
			if(tab[0]->rows == 0)
				break;

			virg_tablet_meta *temp = res[0];
			virg_vm_allocresult(v, vm_, res, res[0]);
			virg_tablet_unlock(v, temp->id);
		}

		// destruct timers
		hipEventDestroy(start);
		hipEventDestroy(data);
		hipEventDestroy(kernel);
		hipEventDestroy(results);
	}

	// wait for all cuda operations to finish
	hipDeviceSynchronize();

	return VIRG_SUCCESS;
}

/// shared memory used for reductions
__shared__ int reduct[512];
/// shared memory location for the start of this threadblock's result area
__shared__ unsigned bstart;
/// shared memory variable for the number of rows output by this threadblock
__shared__ unsigned block;

/// where in the order this threadblock writes its results
__shared__ unsigned shared_blockorder;
///	how many rows have been written in the first result block
__shared__ unsigned thisblockwritten;
///	how many rows have been written in the second result block
__shared__ unsigned nextblockwritten;


#define OPARGS (															   \
	virg_op op,																   \
	virg_vm_context &context, 												   \
	virg_tablet_meta *meta_tab,												   \
	virg_tablet_meta *meta_res,												   \
	void *tab,																   \
	virg_tablet_meta *res,													   \
	void *scratch,															   \
	int &valid,																   \
	unsigned &pc,															   \
	unsigned &pc_wait)

__device__ __forceinline__ void op_Column OPARGS
{
	char *p = (char*)tab + meta_tab->fixed_block;
	unsigned row = blockIdx.x * blockDim.x + threadIdx.x;

	p += meta_tab->fixed_offset[op.p2] + meta_tab->fixed_stride[op.p2] * row;

	switch(meta_tab->fixed_stride[op.p2]) {
		case 4: context.reg[op.p1].i = *((int*)p); break;
		case 8: context.reg[op.p1].d = *((double*)p); break;
		case 1: context.reg[op.p1].c = *p; break;
	}																
	context.type[op.p1] = meta_tab->fixed_type[op.p2];
	context.stride[op.p1] = meta_tab->fixed_stride[op.p2];
}

__device__ __forceinline__ void op_Rowid OPARGS
{
	unsigned row = blockIdx.x * blockDim.x + threadIdx.x;
	char *p = (char*)tab + meta_tab->key_block + meta_tab->key_stride * row;

	switch(meta_tab->fixed_stride[op.p2]) {
		case 4: context.reg[op.p1].i = *((int*)p); break;
		case 8: context.reg[op.p1].d = *((double*)p); break;
		case 1: context.reg[op.p1].c = *p; break;
	}																
	context.type[op.p1] = meta_tab->fixed_type[op.p2];
	context.stride[op.p1] = meta_tab->fixed_stride[op.p2];
}

__device__ __forceinline__ void op_Integer OPARGS
{
	context.reg[op.p1].i = op.p2;
	context.type[op.p1] = VIRG_INT;
	context.stride[op.p1] = sizeof(int);
}

__device__ __forceinline__ void op_Float OPARGS
{
	context.reg[op.p1].f = op.p4.f;
	context.type[op.p1] = VIRG_FLOAT;
	context.stride[op.p1] = sizeof(float);
}

__device__ __forceinline__ void op_Invalid OPARGS
{
	valid = 0;
}

/**
 * A macro to compare identically typed register values and manipulate each
 * thread's program counter based on the result of this comparison. This is
 * implemented as a macro so that the comparison operator can be easily changed,
 * for example REGCMP(<=), used by the Le opcode.
 */
#define REGCMP(cmpop)														   \
	int x = 0;																   \
	switch(context.type[op.p1]) {											   \
		case VIRG_INT:														   \
			x = (context.reg[op.p1].i cmpop context.reg[op.p2].i);		   	   \
			break;															   \
		case VIRG_FLOAT:													   \
			x = (context.reg[op.p1].f cmpop context.reg[op.p2].f);			   \
			break;															   \
		case VIRG_INT64:													   \
			x = (context.reg[op.p1].li cmpop context.reg[op.p2].li);		   \
			break;															   \
		case VIRG_DOUBLE:													   \
			x = (context.reg[op.p1].d cmpop context.reg[op.p2].d);			   \
			break;															   \
		case VIRG_CHAR:														   \
			x = (context.reg[op.p1].c cmpop context.reg[op.p2].c);			   \
			break;															   \
	}																		   \
	if(x) {																	   \
		if(valid)															   \
			valid = op.p4.i;												   \
		pc_wait = op.p3 - pc - 1;											   \
	}


__device__ __forceinline__ void op_Neq	OPARGS { REGCMP(!=) }
__device__ __forceinline__ void op_Gt	OPARGS { REGCMP(>)  }
__device__ __forceinline__ void op_Ge	OPARGS { REGCMP(>=) }
__device__ __forceinline__ void op_Lt	OPARGS { REGCMP(<)  }
__device__ __forceinline__ void op_Le	OPARGS { REGCMP(<=) }
__device__ __forceinline__ void op_And	OPARGS { REGCMP(&&) }
__device__ __forceinline__ void op_Or	OPARGS { REGCMP(||) }

__device__ __forceinline__ void op_Eq	OPARGS
{
	int x = 0;														
	switch(context.type[op.p1]) {									
		case VIRG_INT:												
			x = (context.reg[op.p1].i == context.reg[op.p2].i);	
			break;													
		case VIRG_FLOAT:
		{
			float f = context.reg[op.p1].f - context.reg[op.p2].f;
			x = (f <= VIRG_FLOAT_ERROR && f >= -VIRG_FLOAT_ERROR);
			break;													
		}
		case VIRG_INT64:											
			x = (context.reg[op.p1].li == context.reg[op.p2].li);
			break;													
		case VIRG_DOUBLE:											
		{
			float d = context.reg[op.p1].f - context.reg[op.p2].f;
			x = (d < VIRG_FLOAT_ERROR && d > -VIRG_FLOAT_ERROR);
			break;													
		}
		case VIRG_CHAR:												
			x = (context.reg[op.p1].c == context.reg[op.p2].c);	
			break;													
	}																
	if(x) {															
		if(valid)													
			valid = op.p4.i;										
		pc_wait = op.p3 - pc - 1;									
	}
}

__device__ __forceinline__ void op_Not OPARGS
{
	int x = 0;
	switch(context.type[op.p1]) {
		case VIRG_INT:
			x = (context.reg[op.p1].i ? 1 : 0);
			break;
		case VIRG_FLOAT:
			x = (context.reg[op.p1].f ? 1 : 0);
			break;
		case VIRG_INT64:
			x = (context.reg[op.p1].li ? 1 : 0);
			break;
		case VIRG_DOUBLE:
			x = (context.reg[op.p1].d ? 1 : 0);
			break;
		case VIRG_CHAR:
			x = (context.reg[op.p1].c ? 1 : 0);
			break;
	}
	if(!x) {
		if(valid)
			valid = op.p4.i;
		pc_wait = op.p3 - pc - 1;
	}
}

/** A macro to perform a mathematical operation of the form
 * reg[p1] = reg[p2] operator reg[p3]. Like regcmp, this is used so that
 * multiple opcodes can use this code and easily change the math operator, as in
 * MATHOP(+).
 */
#define MATHOP(mop)															   \
	switch(context.type[op.p2]) {											   \
		case VIRG_INT:														   \
			context.reg[op.p1].i =											   \
				 (context.reg[op.p2].i mop context.reg[op.p3].i);			   \
			break;															   \
		case VIRG_FLOAT:													   \
			context.reg[op.p1].f =											   \
				(context.reg[op.p2].f mop context.reg[op.p3].f);			   \
			break;															   \
		case VIRG_INT64:													   \
			context.reg[op.p1].li =											   \
				(context.reg[op.p2].li mop context.reg[op.p3].li);			   \
			break;															   \
		case VIRG_DOUBLE:													   \
			context.reg[op.p1].d = 											   \
				(context.reg[op.p2].d mop context.reg[op.p3].d);			   \
			break;															   \
		case VIRG_CHAR:														   \
			context.reg[op.p1].c =											   \
				(context.reg[op.p2].c mop context.reg[op.p3].c);			   \
			break;															   \
	}																		   \
	context.type[op.p1] = context.type[op.p2];							       \
	context.stride[op.p1] = context.stride[op.p2];

__device__ __forceinline__ void op_Add OPARGS { MATHOP(+) }
__device__ __forceinline__ void op_Sub OPARGS { MATHOP(-) }
__device__ __forceinline__ void op_Mul OPARGS { MATHOP(*) }
__device__ __forceinline__ void op_Div OPARGS { MATHOP(/) }


/**
 * A convenience macro for castng a register from one type to another
 */
#define CASTREG(reg_, destkey, t, srckey)									   \
	context.reg[reg_].destkey = (t) context.reg[reg_].srckey;


__device__ __forceinline__ void op_Cast OPARGS
{
	switch(op.p1) {
		case VIRG_INT:
			switch(context.type[op.p2]) {
				case VIRG_FLOAT:
					CASTREG(op.p2, i, int, f);
					break;
				case VIRG_INT64:
					CASTREG(op.p2, i, int, li);
					break;
				case VIRG_DOUBLE:
					CASTREG(op.p2, i, int, d);
					break;
				case VIRG_CHAR:
					CASTREG(op.p2, i, int, c);
					break;
			}
			context.stride[op.p2] = sizeof(int);
			break;
		case VIRG_FLOAT:
			switch(context.type[op.p2]) {
				case VIRG_INT:
					CASTREG(op.p2, f, float, i);
					break;
				case VIRG_INT64:
					CASTREG(op.p2, f, float, li);
					break;
				case VIRG_DOUBLE:
					CASTREG(op.p2, f, float, d);
					break;
				case VIRG_CHAR:
					CASTREG(op.p2, f, float, c);
					break;
			}
			context.stride[op.p2] = sizeof(int);
			break;
		case VIRG_INT64:
			switch(context.type[op.p2]) {
				case VIRG_INT:
					CASTREG(op.p2, li, long long int, i);
					break;
				case VIRG_FLOAT:
					CASTREG(op.p2, li, long long int, f);
					break;
				case VIRG_DOUBLE:
					CASTREG(op.p2, li, long long int, d);
					break;
				case VIRG_CHAR:
					CASTREG(op.p2, li, long long int, c);
					break;
			}
			context.stride[op.p2] = sizeof(int);
			break;
		case VIRG_DOUBLE:
			switch(context.type[op.p2]) {
				case VIRG_INT:
					CASTREG(op.p2, d, double, i);
					break;
				case VIRG_FLOAT:
					CASTREG(op.p2, d, double, f);
					break;
				case VIRG_INT64:
					CASTREG(op.p2, d, double, li);
					break;
				case VIRG_CHAR:
					CASTREG(op.p2, d, double, c);
					break;
			}
			context.stride[op.p2] = sizeof(int);
			break;
		case VIRG_CHAR:
			switch(context.type[op.p2]) {
				case VIRG_INT:
					CASTREG(op.p2, c, char, i);
					break;
				case VIRG_FLOAT:
					CASTREG(op.p2, c, char, f);
					break;
				case VIRG_INT64:
					CASTREG(op.p2, c, char, li);
					break;
				case VIRG_CHAR:
					CASTREG(op.p2, c, char, d);
					break;
			}
			context.stride[op.p2] = sizeof(int);
			break;
	}
	context.type[op.p2] = (virg_t)op.p1;
}


__device__ __forceinline__ void op_Result OPARGS
{
	/**
	 * To manage outputting result rows, every thread atomically increments a
	 * shared variable to determine how many result rows there will be, then the
	 * first thread in the block atomically adds this number to a global
	 * variable, with appropriate threadblock synchronization in between these
	 * calls. This is more efficient than a scan operation in this case, since
	 * we don't care about the order of the rows and shared memory atomic
	 * operations are actually fairly efficient when compared to reduction
	 * operations, which tend to cause shared memory bank conflicts.
	 */
	unsigned place;

	__syncthreads();

	// if this is a valid row, update the shared variable
	if(valid)
		place = atomicAdd(&block, 1);

	__syncthreads();

	unsigned num_rows = block;

	// the first thread in the block updates the global variable
	if(threadIdx.x == 0) {
		if(scratch != NULL) // mapped
			bstart = atomicAdd(&row_counter, block);
		else {
			bstart = atomicAdd(&res->rows, block);
		}
	}

	__syncthreads();

	// TODO check for result tablet overflow

	unsigned block_start = bstart;
	//block = 0;
	unsigned j;
	char *p;

	// if this is a result row
	if(valid)
		// for every register to be output for this result row
		for(j = op.p1; j < op.p1 + op.p2; j++) {
			// register/column stride
			unsigned stride = context.stride[j];
			unsigned col_location = stride * (block_start + place);

			// if not mapped, write to the result tablet
			if(scratch == NULL || VIRG_NOTWOSTEP)
				p = (char*)res->base_ptr + meta_res->fixed_block +
					meta_res->fixed_offset[j - op.p1] + col_location;

			// if mapped, write to the scratch memory area for buffering before
			// sending across the PCI bus
			else
				p = (char*)scratch + meta_res->fixed_block +
					meta_res->fixed_offset[j - op.p1] + col_location;

			//printf("write row %u\n", place);

			// switch the write based on the variable stride
			switch(stride) {
				case 4:
					((int*)p)[0] = context.reg[j].i;
					break;
				case 8:
					((double*)p)[0] = context.reg[j].d;
					break;
				case 1:
					p[0] = context.reg[j].c;
					break;
			}
		}


	/**
	 * If mapped memory is being used, result rows are written back to main
	 * memory in a two step process. This both reduces the number of accesses
	 * that have to cross the PCI bus, and ensures that the results are being
	 * coalesced properly, as the coalescing rules for mapped memory appear to
	 * be slightly more strict than for GPU global memory. First, result rows
	 * are written to GPU global memory, exactly how they would be if serial
	 * execution and memory transfers were being used. The results are divided
	 * into threadblock-sized blocks, and once we are sure that the results are
	 * written to global memory, we increment the global counters for the blocks
	 * that we have written to. After incrementing, we check to see if the
	 * blocks we have written to are completely filled with rows. If so, every
	 * thread in the threadblock transfers a row from the result block to main
	 * memory, thus maximizing efficiency and coalescing. Tests show that this
	 * is a very good way of managing result transfers of this kind, since
	 * transfers back to main memory proceed efficiently but are overlapped with
	 * kernel execution.
	 */

	// if we are using mapped memory
	if(scratch != NULL && !VIRG_NOTWOSTEP)
	{
		// make sure our result writes have reached global memory
		__threadfence();
		__syncthreads();

		// only do this in the first cuda thread
		if(threadIdx.x == 0) {
			shared_blockorder = atomicAdd(&threadblock_order, 1);
			
			if(num_rows > 0) {
				unsigned *threadswritten = (unsigned*)((char*)scratch +
					VIRG_TABLET_SIZE);
				unsigned result_blockid = block_start / VIRG_THREADSPERBLOCK;//blockDim.x;

				unsigned blockbreak = ((block_start + num_rows - 1) &
					VIRG_THREADSPERBLOCK_MASK);
				unsigned x, y;
			
				// if the results span over two different thread block areas
				if((blockbreak > (block_start & VIRG_THREADSPERBLOCK_MASK)) &&
					(block_start + num_rows != blockbreak))
				{
					// increment the rows that have been written for both
					unsigned thisnewrows = blockbreak - block_start;
					x = atomicAdd(&threadswritten[result_blockid], thisnewrows);
					thisblockwritten = x + thisnewrows;

					unsigned nextnewrows = block_start + num_rows - blockbreak;
					y = atomicAdd(&threadswritten[result_blockid + 1],
						nextnewrows);
					nextblockwritten = y + nextnewrows;
				}
				// otherwise increment the rows that have been written for only
				// this result threadblock area
				else {
					x = atomicAdd(&threadswritten[result_blockid], num_rows);
					thisblockwritten = x + num_rows;
					nextblockwritten = 0;
				}
			}
		}

		__syncthreads();
	
		// if an entire threadblock-sized area has been filled with result rows
		if(num_rows > 0 && thisblockwritten == VIRG_THREADSPERBLOCK)
		{
			unsigned aligned_start = block_start & VIRG_THREADSPERBLOCK_MASK;

			// do coalesced writes from global memory to mapped main memory of
			// this block of results
			for(j = op.p1; j < op.p1 + op.p2; j++) {
				unsigned stride = context.stride[j];

				p = (char*)res->base_ptr + meta_res->fixed_block +
					meta_res->fixed_offset[j - op.p1] + stride * (aligned_start + threadIdx.x);
				char *p_src = (char*)scratch + meta_res->fixed_block +
					meta_res->fixed_offset[j - op.p1] + stride * (aligned_start + threadIdx.x);

				switch(stride) {
					case 4:
						((int*)p)[0] = ((int*)p_src)[0];
						break;
					case 8:
						((int*)p)[0] = ((int*)p_src)[0];
						break;
					case 1:
						if(threadIdx.x < VIRG_THREADSPERBLOCK / 4)
							((int*)p)[0] = ((int*)p_src)[0];
						break;
				}
			}
		}

		if(shared_blockorder == gridDim.x - 1)
		{
			unsigned aligned_start = row_counter & VIRG_THREADSPERBLOCK_MASK;

			// do coalesced writes from global memory to mapped main memory of
			// this block of results
			for(j = op.p1; j < op.p1 + op.p2; j++) {
				unsigned stride = context.stride[j];

				p = (char*)res->base_ptr + meta_res->fixed_block +
					meta_res->fixed_offset[j - op.p1] + stride * (aligned_start + threadIdx.x);
				char *p_src = (char*)scratch + meta_res->fixed_block +
					meta_res->fixed_offset[j - op.p1] + stride * (aligned_start + threadIdx.x);

				switch(stride) {
					case 4:
						((int*)p)[0] = ((int*)p_src)[0];
						break;
					case 8:
						((int*)p)[0] = ((int*)p_src)[0];
						break;
					case 1:
						if(threadIdx.x < VIRG_THREADSPERBLOCK / 4)
							((int*)p)[0] = ((int*)p_src)[0];
						break;
				}
			}
		}

		// if a second thread-block sized area has been filled with result rows
		if(num_rows > 0 && nextblockwritten == VIRG_THREADSPERBLOCK)
		{
			unsigned aligned_start = (block_start + num_rows) & VIRG_THREADSPERBLOCK_MASK;
			// do coalesced writes from global memory to mapped main memory of
			// this block of results
			for(j = op.p1; j < op.p1 + op.p2; j++) {
				unsigned stride = context.stride[j];

				p = (char*)res->base_ptr + meta_res->fixed_block +
					meta_res->fixed_offset[j - op.p1] + stride * (aligned_start + threadIdx.x);
				char *p_src = (char*)scratch + meta_res->fixed_block +
					meta_res->fixed_offset[j - op.p1] + stride * (aligned_start + threadIdx.x);

				switch(stride) {
					case 4:
						((int*)p)[0] = ((int*)p_src)[0];
						break;
					case 8:
						((int*)p)[0] = ((int*)p_src)[0];
						break;
					case 1:
						if(threadIdx.x < VIRG_THREADSPERBLOCK / 4)
							((int*)p)[0] = ((int*)p_src)[0];
						break;
				}
			}
		}
	} // if we are using mapped memory


}


/**
 * @ingroup vm
 * @brief CUDA virtual machine kernel
 *
 * This function executes a virtual machine context on a data tablet in
 * parallel. Opcodes are are accessed using a switch statement, since there is
 * no support for indirect jumping on current NVIDIA hardware.
 *
 * @param tab_slot		The GPU constant memory slot containing the data
 * tablet's meta information
 * @param res_slot		The GPU constant memory slot containing the result
 * tablet's meta information
 * @param tab_			Pointer to the data tablet
 * @param res_			Pointer to the result tablet
 * @param start_row		The row at which to start processing the data tablet
 * @param num_rows		The number of rows to process from the data tablet, 0
 * for as many as possible
 * @param scratch		Buffer tablet used to store intermediate results in
 * global memory before moving them to mapped main memory, only set for mapped
 * memory execution
 * @return VIRG_SUCCESS or VIRG_FAIL depending on errors during the function
 * call
 */
__global__ void virginia_gpu(
	unsigned tab_slot,
	unsigned res_slot,
	void* tab_,
	void* res_,
	unsigned start_row,
	unsigned num_rows,
	void *scratch)
{
	// tablet pointers
	virg_tablet_meta *res = (virg_tablet_meta*)res_;
	virg_tablet_meta *meta_tab = &meta[tab_slot];
	virg_tablet_meta *meta_res = &meta[res_slot];

	// misc kernel variables
	unsigned pc 		= vm.pc;
	unsigned pc_wait 	= 0;
	int	valid 			= 1;
	unsigned row		= blockIdx.x * blockDim.x + threadIdx.x;
	if(threadIdx.x == 0)
		block = 0;
	virg_vm_context	context;

	// if we've reached the end of the data tablet or the number of rows we're
	// supposed to process in this kernel launch then this row is not valid,
	// otherwise go to the row calculated with the thread id and block id
	if(row >= meta_tab->rows || (row >= num_rows && num_rows != 0))
		valid = 0;
	else
		row += start_row;

//	int op = vm.stmt[pc].op;
//	__asm(".global .u32 jmptbl[2] = {op_Column, op_Integer};");
//	__asm("bra %%op, jmptbl;");
	while(1)
	{
		// if this thread has diverged and is waiting at a later opcode, then
		// don't switch on the current opcode
		if(pc_wait > 0)
			pc_wait--;
		// otherwise switch on the current global opcode
		else {

#define ARG (vm.stmt[pc], context, meta_tab, meta_res, tab_, res, scratch, valid, pc, pc_wait)

			switch(vm.stmt[pc].op) {
				case OP_Column	: op_Column		ARG; break;
				case OP_Rowid	: op_Rowid		ARG; break;
				case OP_Result	: op_Result		ARG; break;
				case OP_Invalid	: op_Invalid	ARG; break;
				case OP_Integer	: op_Integer	ARG; break;
				case OP_Float	: op_Float		ARG; break;
				case OP_Converge: return;
				case OP_Le		: op_Le			ARG; break;
				case OP_Lt		: op_Lt			ARG; break;
				case OP_Ge		: op_Ge			ARG; break;
				case OP_Gt		: op_Gt			ARG; break;
				case OP_Eq		: op_Eq			ARG; break;
				case OP_Neq		: op_Neq		ARG; break;
				case OP_Add 	: op_Add 		ARG; break;
				case OP_Sub		: op_Sub		ARG; break;
				case OP_Mul		: op_Mul		ARG; break;
				case OP_Div		: op_Div		ARG; break;
				case OP_And		: op_And		ARG; break;
				case OP_Or		: op_Or			ARG; break;
				case OP_Not		: op_Not		ARG; break;
				case OP_Cast	: op_Cast		ARG; break;
			}
		}

		pc++;
	}
}

