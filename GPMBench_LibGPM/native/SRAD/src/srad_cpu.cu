#include "hip/hip_runtime.h"
// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include "srad.h"
#include "libgpm.cuh"

// includes, project
#include <hip/hip_runtime.h>

// includes, kernels
#include "srad_kernel.cu"

void random_matrix(float *I, int rows, int cols);
void runTest( int argc, char** argv);
void usage(int argc, char **argv)
{
	fprintf(stderr, "Usage: %s <rows> <cols> <y1> <y2> <x1> <x2> <lamda> <no. of iter> <no. of thd>\n", argv[0]);
	fprintf(stderr, "\t<rows>   - number of rows\n");
	fprintf(stderr, "\t<cols>    - number of cols\n");
	fprintf(stderr, "\t<y1> 	 - y1 value of the speckle\n");
	fprintf(stderr, "\t<y2>      - y2 value of the speckle\n");
	fprintf(stderr, "\t<x1>       - x1 value of the speckle\n");
	fprintf(stderr, "\t<x2>       - x2 value of the speckle\n");
	fprintf(stderr, "\t<lamda>   - lambda (0,1)\n");
	fprintf(stderr, "\t<no. of iter>   - number of iterations\n");
	fprintf(stderr, "\t<no. of thd>   - number of CPU threads\n");
	
	exit(1);
}

double kernel_time = 0, persist_time = 0, ddio_time = 0;

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main( int argc, char** argv) 
{
  printf("WG size of kernel = %d X %d\n", BLOCK_SIZE, BLOCK_SIZE);
    runTest( argc, argv);

    return EXIT_SUCCESS;
}

__global__ void setMemory(float *arr, size_t size, float value)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    for(int i = id; i < size; i += blockDim.x * gridDim.x)
        arr[i] = value;
}


void
runTest( int argc, char** argv) 
{
    long long rows, cols, size_I, size_R, niter = 10, iter, nthread;
    float *I, *J, lambda, q0sqr, sum, sum2, tmp, meanROI,varROI ;

	float Jc, G2, L, num, den, qsqr;
	int *iN,*iS,*jE,*jW, k;
	float *dN,*dS,*dW,*dE;
	float cN,cS,cW,cE,D;
    float *J_cuda_out;
    float *C_cuda;

/*#ifdef GPU
	
	float *J_cuda;
    float *C_cuda;
	float *E_C, *W_C, *N_C, *S_C;

#endif
*/

	unsigned long long r1, r2, c1, c2;
	float *c;
    
	
 
	if (argc >= 10)
	{
		rows = atoi(argv[1]);  //number of rows in the domain
		cols = atoi(argv[2]);  //number of cols in the domain
		if ((rows%16!=0) || (cols%16!=0)){
		fprintf(stderr, "rows and cols must be multiples of 16\n");
		exit(1);
		}
		r1   = atoi(argv[3]);  //y1 position of the speckle
		r2   = atoi(argv[4]);  //y2 position of the speckle
		c1   = atoi(argv[5]);  //x1 position of the speckle
		c2   = atoi(argv[6]);  //x2 position of the speckle
		lambda = atof(argv[7]); //Lambda value
		niter = atoi(argv[8]); //number of iterations
		if(argc >= 10)
			nthread = atoi(argv[9]);
		else
			nthread = 32;
		
	}
    else{
	usage(argc, argv);
    }



	size_I = cols * rows;
    size_R = (r2-r1+1)*(c2-c1+1);   

	I = (float *)malloc( size_I * sizeof(float) );
    J = (float *)malloc( size_I * sizeof(float) );
	c  = (float *)malloc(sizeof(float)* size_I) ;


    const char *path_j = "persist_j.dat";
    const char *path_c = "persist_c.dat";
    const char *path_e_c = "persist_e_c.dat";
    const char *path_w_c = "persist_w_c.dat";
    const char *path_s_c = "persist_s_c.dat";
    const char *path_n_c = "persist_n_c.dat";
    const char *path_j_out = "persist_j_out.dat";

    printf("Randomizing the input matrix\n");
    //Generate a random matrix
    random_matrix(I, rows, cols);
    for (int k = 0;  k < size_I; k++ ) {
        J[k] = (float)exp(I[k]);
    }

    iN = (int *)malloc(sizeof(unsigned int*) * rows) ;
    iS = (int *)malloc(sizeof(unsigned int*) * rows) ;
    jW = (int *)malloc(sizeof(unsigned int*) * cols) ;
    jE = (int *)malloc(sizeof(unsigned int*) * cols) ;    

    size_t len = sizeof(float) * size_I;   
    C_cuda = (float*) gpm_map_file(path_c, len, true);
    dE    = (float*) gpm_map_file(path_e_c, len, true);
    dW    = (float*) gpm_map_file(path_w_c, len, true);
    dN    = (float*) gpm_map_file(path_n_c, len, true);
    dS    = (float*) gpm_map_file(path_s_c, len, true);
    J_cuda_out = (float*) gpm_map_file(path_j_out, len, true);
    

    for (int i=0; i< rows; i++) {
        iN[i] = i-1;
        iS[i] = i+1;
    }    
    for (int j=0; j< cols; j++) {
        jW[j] = j-1;
        jE[j] = j+1;
    }
    iN[0]    = 0;
    iS[rows-1] = rows-1;
    jW[0]    = 0;
    jE[cols-1] = cols-1;

//#endif

/*
#ifdef GPU

	//Allocate device memory
    hipMalloc((void**)& J_cuda, sizeof(float)* size_I);
    hipMalloc((void**)& C_cuda, sizeof(float)* size_I);
	hipMalloc((void**)& E_C, sizeof(float)* size_I);
	hipMalloc((void**)& W_C, sizeof(float)* size_I);
	hipMalloc((void**)& S_C, sizeof(float)* size_I);
	hipMalloc((void**)& N_C, sizeof(float)* size_I);

	
#endif 
*/

	printf("Randomizing the input matrix\n");
	//Generate a random matrix
	random_matrix(I, rows, cols);

    for (int k = 0;  k < size_I; k++ ) {
     	J[k] = (float)exp(I[k]) ;
    }
	printf("Start the SRAD main loop\n");
 for (iter=0; iter< niter; iter++){     
		sum=0; sum2=0;
        for (int i=r1; i<=r2; i++) {
            for (int j=c1; j<=c2; j++) {
                tmp   = J[i * cols + j];
                sum  += tmp ;
                sum2 += tmp*tmp;
            }
        }
        meanROI = sum / size_R;
        varROI  = (sum2 / size_R) - meanROI*meanROI;
        q0sqr   = varROI / (meanROI*meanROI);

        
    	auto start = std::chrono::high_resolution_clock::now(); 
    	#pragma omp parallel for num_threads(nthread)
		for (int i = 0 ; i < rows ; i++) {
            for (int j = 0; j < cols; j++) { 
		
				k = i * cols + j;
				Jc = J[k];
 
				// directional derivates
                dN[k] = J[iN[i] * cols + j] - Jc;
                dS[k] = J[iS[i] * cols + j] - Jc;
                dW[k] = J[i * cols + jW[j]] - Jc;
                dE[k] = J[i * cols + jE[j]] - Jc;
			    pmem_flush(&dN[k], sizeof(float));
			    pmem_flush(&dS[k], sizeof(float));
			    pmem_flush(&dW[k], sizeof(float));
			    pmem_flush(&dE[k], sizeof(float));
                pmem_drain();
                G2 = (dN[k]*dN[k] + dS[k]*dS[k] 
                    + dW[k]*dW[k] + dE[k]*dE[k]) / (Jc*Jc);

   		        L = (dN[k] + dS[k] + dW[k] + dE[k]) / Jc;

				num  = (0.5*G2) - ((1.0/16.0)*(L*L)) ;
                den  = 1 + (.25*L);
                qsqr = num/(den*den);
 
                // diffusion coefficent (equ 33)
                den = (qsqr-q0sqr) / (q0sqr * (1+q0sqr)) ;
                C_cuda[k] = 1.0 / (1.0+den) ;
                
                // saturate diffusion coefficent
                if (C_cuda[k] < 0) {C_cuda[k] = 0;}
                else if (C_cuda[k] > 1) {C_cuda[k] = 1;}
			}
		}
		
    	#pragma omp parallel for num_threads(nthread)
        for (int i = 0; i < rows; i++) {
            for (int j = 0; j < cols; j++) {        

                // current index
                k = i * cols + j;
                
                // diffusion coefficent
					cN = c[k];
					cS = c[iS[i] * cols + j];
					cW = c[k];
					cE = c[i * cols + jE[j]];

                // divergence (equ 58)
                D = cN * dN[k] + cS * dS[k] + cW * dW[k] + cE * dE[k];
                
                // image update (equ 61)
                J_cuda_out[k] = J[k] + 0.25*lambda*D;
            }
		}        
		memcpy(J, J_cuda_out, sizeof(float) * size_I);
		kernel_time += (double)std::chrono::duration_cast<std::chrono::microseconds>(std::chrono::high_resolution_clock::now() - start).count() / 1000.0;    
		printf("Iteration %d, time so far: %f\n", iter, kernel_time);

	/*
	#ifdef GPU

		//Currently the input size must be divided by 16 - the block size
		int block_x = cols/BLOCK_SIZE ;
		int block_y = rows/BLOCK_SIZE ;

		dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
		dim3 dimGrid(block_x , block_y);
		

		//Copy data from main memory to device memory
		hipMemcpy(J_cuda, J, sizeof(float) * size_I, hipMemcpyHostToDevice);

		//Run kernels
		srad_cuda_1<<<dimGrid, dimBlock>>>(E_C, W_C, N_C, S_C, J_cuda, C_cuda, cols, rows, q0sqr); 
		srad_cuda_2<<<dimGrid, dimBlock>>>(E_C, W_C, N_C, S_C, J_cuda, C_cuda, cols, rows, lambda, q0sqr); 

		//Copy data from device memory to main memory
		hipMemcpy(J, J_cuda, sizeof(float) * size_I, hipMemcpyDeviceToHost);

	#endif
	*/   
	}

    hipDeviceSynchronize();

#ifdef OUTPUT
    //Printing output	
		printf("Printing Output:\n"); 
    for( int i = 0 ; i < rows ; i++){
		for ( int j = 0 ; j < cols ; j++){
         printf("%.5f ", J[i * cols + j]); 
		}	
     printf("\n"); 
   }
#endif 

	printf("runtime: %f\n", kernel_time);
	printf("Computation Done\n");

	free(I);
	free(J);
//#ifdef CPU
	free(iN); free(iS); free(jW); free(jE);
    //free(dN); free(dS); free(dW); free(dE);
//#endif
/*
#ifdef GPU
    hipFree(C_cuda);
	hipFree(J_cuda);
	hipFree(E_C);
	hipFree(W_C);
	hipFree(N_C);
	hipFree(S_C);
#endif 
*/
	free(c);
  
}


void random_matrix(float *I, int rows, int cols){
    
	srand(7);
	
	for( int i = 0 ; i < rows ; i++){
		for ( int j = 0 ; j < cols ; j++){
		 I[i * cols + j] = rand()/(float)RAND_MAX ;
		}
	}

}

